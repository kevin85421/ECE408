#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_size = ceil((1.0 * Width_out)/TILE_WIDTH);
    int m = blockIdx.x;
    int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x;
    int b = blockIdx.z;

    int mh1, mw1, mh2, mw2;

    // float acc = 0.0f;
    half2 acc = __floats2half2_rn(0.0f,0.0f);
    if (h < Height_out && w < Width_out) {
        for (int c=0; c < Channel; c++) {
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 0, 0), mask_4d(m, c, 0, 1)), __floats2half2_rn(in_4d(b, c, h + 0, w + 0), in_4d(b, c, h + 0, w + 1)))); 
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 0, 2), mask_4d(m, c, 0, 3)), __floats2half2_rn(in_4d(b, c, h + 0, w + 2), in_4d(b, c, h + 0, w + 3))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 0, 4), mask_4d(m, c, 0, 5)), __floats2half2_rn(in_4d(b, c, h + 0, w + 4), in_4d(b, c, h + 0, w + 5))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 0, 6), mask_4d(m, c, 1, 0)), __floats2half2_rn(in_4d(b, c, h + 0, w + 6), in_4d(b, c, h + 1, w + 0))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 1, 1), mask_4d(m, c, 1, 2)), __floats2half2_rn(in_4d(b, c, h + 1, w + 1), in_4d(b, c, h + 1, w + 2))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 1, 3), mask_4d(m, c, 1, 4)), __floats2half2_rn(in_4d(b, c, h + 1, w + 3), in_4d(b, c, h + 1, w + 4))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 1, 5), mask_4d(m, c, 1, 6)), __floats2half2_rn(in_4d(b, c, h + 1, w + 5), in_4d(b, c, h + 1, w + 6))));   
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 2, 0), mask_4d(m, c, 2, 1)), __floats2half2_rn(in_4d(b, c, h + 2, w + 0), in_4d(b, c, h + 2, w + 1)))); 
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 2, 2), mask_4d(m, c, 2, 3)), __floats2half2_rn(in_4d(b, c, h + 2, w + 2), in_4d(b, c, h + 2, w + 3))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 2, 4), mask_4d(m, c, 2, 5)), __floats2half2_rn(in_4d(b, c, h + 2, w + 4), in_4d(b, c, h + 2, w + 5))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 2, 6), mask_4d(m, c, 3, 0)), __floats2half2_rn(in_4d(b, c, h + 2, w + 6), in_4d(b, c, h + 3, w + 0))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 3, 1), mask_4d(m, c, 3, 2)), __floats2half2_rn(in_4d(b, c, h + 3, w + 1), in_4d(b, c, h + 3, w + 2))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 3, 3), mask_4d(m, c, 3, 4)), __floats2half2_rn(in_4d(b, c, h + 3, w + 3), in_4d(b, c, h + 3, w + 4))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 3, 5), mask_4d(m, c, 3, 6)), __floats2half2_rn(in_4d(b, c, h + 3, w + 5), in_4d(b, c, h + 3, w + 6))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 4, 0), mask_4d(m, c, 4, 1)), __floats2half2_rn(in_4d(b, c, h + 4, w + 0), in_4d(b, c, h + 4, w + 1)))); 
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 4, 2), mask_4d(m, c, 4, 3)), __floats2half2_rn(in_4d(b, c, h + 4, w + 2), in_4d(b, c, h + 4, w + 3))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 4, 4), mask_4d(m, c, 4, 5)), __floats2half2_rn(in_4d(b, c, h + 4, w + 4), in_4d(b, c, h + 4, w + 5))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 4, 6), mask_4d(m, c, 5, 0)), __floats2half2_rn(in_4d(b, c, h + 4, w + 6), in_4d(b, c, h + 5, w + 0))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 5, 1), mask_4d(m, c, 5, 2)), __floats2half2_rn(in_4d(b, c, h + 5, w + 1), in_4d(b, c, h + 5, w + 2))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 5, 3), mask_4d(m, c, 5, 4)), __floats2half2_rn(in_4d(b, c, h + 5, w + 3), in_4d(b, c, h + 5, w + 4))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 5, 5), mask_4d(m, c, 5, 6)), __floats2half2_rn(in_4d(b, c, h + 5, w + 5), in_4d(b, c, h + 5, w + 6))));  
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 6, 0), mask_4d(m, c, 6, 1)), __floats2half2_rn(in_4d(b, c, h + 6, w + 0), in_4d(b, c, h + 6, w + 1)))); 
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 6, 2), mask_4d(m, c, 6, 3)), __floats2half2_rn(in_4d(b, c, h + 6, w + 2), in_4d(b, c, h + 6, w + 3))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, 6, 4), mask_4d(m, c, 6, 5)), __floats2half2_rn(in_4d(b, c, h + 6, w + 4), in_4d(b, c, h + 6, w + 5))));
            acc = __hadd2(acc, __hmul2(__floats2half2_rn(mask_4d(m, c, K-1, K-1), 0.0f), __floats2half2_rn(in_4d(b, c, h + K - 1, w + K - 1), 0.0f)));
        }
        out_4d(b, m, h, w) = __half2float(__hadd(acc.x, acc.y));
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1);
    int input_size = Batch * Channel * Height * Width;
    int mask_size = Map_out * Channel * K * K;

    hipMalloc((void **) device_output_ptr, output_size * sizeof(float));
    hipMalloc((void **) device_input_ptr, input_size * sizeof(float));
    hipMalloc((void **) device_mask_ptr, mask_size * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float * __restrict__ device_output, const float * __restrict__ device_input, const float * __restrict__ device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int W_out = Width - K + 1;
    int H_out = Height - K + 1;

    int W_size = ceil((1.0 * W_out)/TILE_WIDTH);
    int H_size = ceil((1.0 * H_out)/TILE_WIDTH);
    int Y = H_size * W_size;

    dim3 dimGrid(Map_out, Y, Batch);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1); 
    hipMemcpy(host_output, device_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask); 
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}

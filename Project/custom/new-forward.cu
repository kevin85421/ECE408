#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_size = ceil((1.0 * Width_out)/TILE_WIDTH);
    int m = blockIdx.x;
    int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x;
    int b = blockIdx.z;

    float acc = 0.0f;
    if (h < Height_out && w < Width_out) {
        for (int c=0; c < Channel; c++) {
            for (int p=0; p < K; p++) {
                for (int q=0; q < K; q++)
                    acc += in_4d(b, c, h+p, w+q) * mask_4d(m, c, p, q);
            }
        }
        out_4d(b, m, h, w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1);
    int input_size = Batch * Channel * Height * Width;
    int mask_size = Map_out * Channel * K * K;

    hipMalloc((void **) device_output_ptr, output_size * sizeof(float));
    hipMalloc((void **) device_input_ptr, input_size * sizeof(float));
    hipMalloc((void **) device_mask_ptr, mask_size * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, mask_size * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float * __restrict__ device_output, const float * __restrict__ device_input, const float * __restrict__ device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int W_out = Width - K + 1;
    int H_out = Height - K + 1;

    int W_size = ceil((1.0 * W_out)/TILE_WIDTH);
    int H_size = ceil((1.0 * H_out)/TILE_WIDTH);
    int Y = H_size * W_size;

    dim3 dimGrid(Map_out, Y, Batch);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int output_size = Batch * Map_out * (Height - K + 1) * (Width - K + 1); 
    hipMemcpy(host_output, device_output, output_size * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask); 
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
